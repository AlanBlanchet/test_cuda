#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024 // You can adjust the matrix size as needed

__global__ void matrixMulKernel(float *A, float *B, float *C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n)
    {
        float value = 0;
        for (int k = 0; k < n; ++k)
        {
            value += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = value;
    }
}

extern "C" void matrixMulCUDA(float *h_A, float *h_B, float *h_C)
{
    int size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
